
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <time.h>
#define SIZE 32

typedef struct bmpFileHeaderStruct {
  /* 2 bytes de identificación */
  uint32_t size;        /* Tamaño del archivo */
  uint16_t resv1;       /* Reservado */
  uint16_t resv2;       /* Reservado */
  uint32_t offset;      /* Offset hasta hasta los datos de imagen */
} bmpFileHeader;

typedef struct bmpInfoHeaderStruct {
  uint32_t headersize;  /* Tamaño de la cabecera */
  uint32_t width;       /* Ancho */
  uint32_t height;      /* Alto */
  uint16_t planes;      /* Planos de color (Siempre 1) */
  uint16_t bpp;         /* bits por pixel */
  uint32_t compress;    /* compresion */
  uint32_t imgsize;     /* tamaño de los datos de imagen */
  uint32_t bpmx;        /* Resolucion X en bits por metro */
  uint32_t bpmy;        /* Resolucion Y en bits por metro */
  uint32_t colors;      /* colors used en la paleta */
  uint32_t imxtcolors;  /* Colores importantes. 0 si son todos */
} bmpInfoHeader;


void BW(unsigned char *imgdata, bmpInfoHeader *bInfoHeader) {
    
    float color;
    int x, y;

        //KERNEL
        for(x = 0; x < bInfoHeader->width; x++)
        {
            for(y = 0; y < bInfoHeader->height; y++)
            {
                color += imgdata[x*3 + y*bInfoHeader->width*3 + 0] * 0.114;
                color += imgdata[x*3 + y*bInfoHeader->width*3 + 1] * 0.587;
                color += imgdata[x*3 + y*bInfoHeader->width*3 + 2] * 0.299;
                color /= 3;
                imgdata[x*3 + y*bInfoHeader->width*3 + 0] = color;
                imgdata[x*3 + y*bInfoHeader->width*3 + 1] = color;
                imgdata[x*3 + y*bInfoHeader->width*3 + 2] = color;
            }
        }

        /*imgdata[xx*3 + yy*bInfoHeader->width*3 + 0] = avgB;
        imgdata[xx*3 + yy*bInfoHeader->width*3 + 1] = avgG;
        imgdata[xx*3 + yy*bInfoHeader->width*3 + 2] = avgR;*/

}



__global__ void ConvMatKernel(unsigned char *img_device, unsigned char *img_device2, uint32_t width_image, uint32_t height_image, int j, float *mat) {
    //Hay que pasarle la matriz
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = width_image * row + col;
    float avgB, avgG, avgR;
    int x, y;

    avgB = avgG = avgR = 0;

    if (i < (width_image * height_image)) {          
        for(x = -1; x < 2; x++) {
          if (row == 0 && x == -1) {
              x = 0;
          }
          else if (row == height_image - 1) {
              if (x > 0) break;
          }
          for(y = -1; y < 2; y++) {
              if (col == 0 && y == -1) y = 0;
              if (col == width_image - 1 && y == 1) break;
              avgB += img_device[(col + y)*3 + (x + row) * width_image*3 + 0] * mat[((x + 1) * 3) + y + 1];
              avgG += img_device[(col + y)*3 + (x + row) * width_image*3 + 1] * mat[((x + 1) * 3) + y + 1];
              avgR += img_device[(col + y)*3 + (x + row) * width_image*3 + 2] * mat[((x + 1) * 3) + y + 1];
          }
        }
        img_device2[col*3 + row*width_image*3 + 0] = avgB;
        img_device2[col*3 + row*width_image*3 + 1] = avgG;
        img_device2[col*3 + row*width_image*3 + 2] = avgR;
    }
}

__global__ void blurKernel(unsigned char *img_device, unsigned char *img_device2, uint32_t width_image, uint32_t height_image) {

    int x,y,ile, avgR,avgB,avgG;
    int blurSize = 10;
    avgB = avgG = avgR = 0;
    ile = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = width_image * row + col;

    if (i < (width_image * height_image)) {
        for(x = col; x < width_image && x < col + blurSize; x++)
        {
            for(y = row; y < height_image && y < row + blurSize; y++)
            {
                avgB += img_device2[x*3 + y*width_image*3 + 0];
                avgG += img_device2[x*3 + y*width_image*3 + 1];
                avgR += img_device2[x*3 + y*width_image*3 + 2];
                ile++;
            }
        }
        avgB = avgB / ile;
        avgG = avgG / ile;
        avgR = avgR / ile;

        img_device[col*3 + row*width_image*3 + 0] = avgB;
        img_device[col*3 + row*width_image*3 + 1] = avgG;
        img_device[col*3 + row*width_image*3 + 2] = avgR;
    }
}


//Kernel BW


__global__ void BWkernel(unsigned char *img_device, uint32_t n) {
    float color;
    color = 0.0f;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        color += img_device[i*3 + 0] * 0.114;
        color += img_device[i*3 + 1] * 0.587;
        color += img_device[i*3 + 2] * 0.299;
        color /= 3;
        img_device[i*3 + 0] = color;
        img_device[i*3 + 1] = color;
        img_device[i*3 + 2] = color;
    }
  }

  void CheckCudaError(char sms[], int line);


bmpInfoHeader *createInfoHeader(uint32_t width, uint32_t height, uint32_t ppp) {
	
  bmpInfoHeader *InfoHeader;

  //InfoHeader = malloc(sizeof(InfoHeader));
  if (InfoHeader == NULL) return NULL; 
  InfoHeader->headersize = sizeof(bmpInfoHeader);
  InfoHeader->width = width;
  InfoHeader->height = height;
  InfoHeader->planes = 1;
  InfoHeader->bpp = 24;
  InfoHeader->compress = 0;
  /* 3 bytes por pixel, width*height pixels, el tamaño de las filas ha de ser multiplo de 4 */
  InfoHeader->imgsize = ((width*3 + 3) / 4) * 4 * height;        
  InfoHeader->bpmx = (unsigned) ((double)ppp*100/2.54);
  InfoHeader->bpmy= InfoHeader->bpmx;          /* Misma resolucion vertical y horiontal */
  InfoHeader->colors = 0;
  InfoHeader->imxtcolors = 0;

  return InfoHeader;
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
}




unsigned char *LoadBMP(char *filename, bmpInfoHeader *bInfoHeader, int i) {
  FILE *f;
  bmpFileHeader header;     /* cabecera */
  unsigned char *imgdata_h;   /* datos de imagen */
  unsigned char *imgdata2_h;
  unsigned char *imgdata_d;   
  unsigned char *imgdata2_d;
  uint16_t type;            /* 2 bytes identificativos */
  //Para el kernel
  unsigned int N;
  unsigned int numBytes;
  unsigned int nBlocks, nThreads;
  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;

  f=fopen (filename, "r");
  if (!f) { /* Si no podemos leer, no hay imagen */
    printf("NO se puede abrir el fichero %s\n", filename);
    return NULL;        
  } 

  /* Leemos los dos primeros bytes y comprobamos el formato */
  fread(&type, sizeof(uint16_t), 1, f);
  if (type !=0x4D42) {       
    fclose(f);
    printf("%s NO es una imagen BMP\n", filename);
    return NULL;
  }

  /* Leemos la cabecera del fichero */
  fread(&header, sizeof(bmpFileHeader), 1, f);

  printf("File size: %u\n", header.size);
  printf("Reservado: %u\n", header.resv1);
  printf("Reservado: %u\n", header.resv2);
  printf("Offset:    %u\n", header.offset);

  /* Leemos la cabecera de información del BMP */
  fread(bInfoHeader, sizeof(bmpInfoHeader), 1, f);

  /* Reservamos memoria para la imagen, lo que indique imgsize */
  if (bInfoHeader->imgsize == 0) bInfoHeader->imgsize = ((bInfoHeader->width*3 +3) / 4) * 4 * bInfoHeader->height;
  imgdata_h = (unsigned char*) malloc(bInfoHeader->imgsize);
  imgdata2_h = (unsigned char*) malloc(bInfoHeader->imgsize);
  
  if (imgdata_h == NULL) {
    printf("Fallo en el malloc, del fichero %s\n", filename);
    exit(0);
  }
  /* Nos situamos en donde empiezan los datos de imagen, lo indica el offset de la cabecera de fichero */
  fseek(f, header.offset, SEEK_SET);

  /* Leemos los datos de la imagen, tantos bytes como imgsize */
  fread(imgdata_h, bInfoHeader->imgsize,1, f);
  
  if (i == 1) {
      nThreads = SIZE;
      N = bInfoHeader->imgsize;
      // numero de Blocks en cada dimension 
      uint32_t nBlocksWidth = bInfoHeader->width / nThreads;
      uint32_t nBlocksHeight = bInfoHeader->height / nThreads;

      dim3 dimGrid(nBlocksWidth, nBlocksHeight, 1);
      dim3 dimBlock(nThreads, nThreads, 1);

      hipEventCreate(&E0);
      hipEventCreate(&E1);
      hipEventCreate(&E2);
      hipEventCreate(&E3);

      hipEventRecord(E0, 0);
      hipEventSynchronize(E0);

      hipMalloc((unsigned char**)&imgdata_d, bInfoHeader->imgsize);
      hipMalloc((unsigned char**)&imgdata2_d, bInfoHeader->imgsize);
      CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);

      hipMemcpy(imgdata_d, imgdata_h, bInfoHeader->imgsize, hipMemcpyHostToDevice);
      hipMemcpy(imgdata2_d, imgdata_h, bInfoHeader->imgsize, hipMemcpyHostToDevice);
      CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

      hipEventRecord(E1, 0);
      hipEventSynchronize(E1);
      
      blurKernel<<<dimGrid, dimBlock>>>(imgdata_d, imgdata2_d, bInfoHeader->width, bInfoHeader->height);
      CheckCudaError((char *) "Invocar Kernel", __LINE__);

      hipEventRecord(E2, 0);
      hipEventSynchronize(E2); 

      hipMemcpy(imgdata_h, imgdata_d, bInfoHeader->imgsize, hipMemcpyDeviceToHost);
      CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

      hipEventRecord(E3, 0);
      hipEventSynchronize(E3);
      hipEventElapsedTime(&TiempoTotal,  E0, E3);
      hipEventElapsedTime(&TiempoKernel, E1, E2);
      printf("\nKERNEL BlackAndWhiteFilter\n");
      printf("Dimensiones: %d\n",N);
      printf("nThreads: %dx%d (%d)\n", nThreads, nBlocks, nThreads * nBlocks);
      printf("nBlocks: %d\n", nBlocks);
      printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
      printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
      printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoTotal));
      printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));

      hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
      hipFree(imgdata_d);
      hipFree(imgdata2_d);
  }
  else if (i == 2) {
      // numero de Threads en cada dimension 
    nThreads = SIZE;
    N = bInfoHeader->imgsize;
    // numero de Blocks en cada dimension 
    nBlocks = N / nThreads;
  
	  dim3 dimGrid(nBlocks, 1, 1);
	  dim3 dimBlock(nThreads, 1, 1);
    hipEventCreate(&E0);
    hipEventCreate(&E1);
    hipEventCreate(&E2);
    hipEventCreate(&E3);

    hipEventRecord(E0, 0);
    hipEventSynchronize(E0);

	  hipMalloc((unsigned char**)&imgdata_d, bInfoHeader->imgsize);
    CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);
	  //cudaMalloc((unsigned char**)&imgdata2_d, bInfoHeader->imgsize);
	  hipMemcpy(imgdata_d, imgdata_h, bInfoHeader->imgsize, hipMemcpyHostToDevice);
    CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

	  //cudaMemcpy(imgdata2_d, imgdata2_h, bInfoHeader->imgsize, cudaMemcpyHostToDevice);
    hipEventRecord(E1, 0);
    hipEventSynchronize(E1);
      
	  BWkernel<<<nBlocks, nThreads>>>(imgdata_d, (bInfoHeader->width * bInfoHeader->height));
    CheckCudaError((char *) "Invocar Kernel", __LINE__);

      //BW(imgdata_h, bInfoHeader);
    hipEventRecord(E2, 0);
    hipEventSynchronize(E2); 

	  hipMemcpy(imgdata_h, imgdata_d, bInfoHeader->imgsize, hipMemcpyDeviceToHost);
    CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

    hipEventRecord(E3, 0);
    hipEventSynchronize(E3);
    
    hipEventElapsedTime(&TiempoTotal,  E0, E3);
    hipEventElapsedTime(&TiempoKernel, E1, E2);
    printf("\nKERNEL BlackAndWhiteFilter\n");
    printf("Dimensiones: %d\n",N);
    printf("nThreads: %dx%d (%d)\n", nThreads, nBlocks, nThreads * nBlocks);
    printf("nBlocks: %d\n", nBlocks);
    printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
    printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
    printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoTotal));
    printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));

    hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
    hipFree(imgdata_d);
      //printf("Black&White took %f seconds to execute \n", time_taken);
  }
  else if (i == 3) {
      float mat[9];
      float *mat_d;
      //BW(imgdata_h, bInfoHeader);
      int j = 3;
      if (j == 1) {
          mat[0] = -1.;
          mat[1] = -1.;
          mat[2] = -1.;
          mat[3] = -1.;
          mat[4] = 8.;
          mat[5] = -1.;
          mat[6] = -1.;
          mat[7] = -1.;
          mat[8] = -1.;
      }
      else if (j == 2) {
          mat[0] = 0.;
          mat[1] = -1.;
          mat[2] = 0.;
          mat[3] = -1.;
          mat[4] = 5.;
          mat[5] = -1.;
          mat[6] = 0.;
          mat[7] = -1.;
          mat[8] = 0.;
      }
      else if (j == 3) {
          mat[0] = 1./16.;
          mat[1] = 2./16.;
          mat[2] = 1./16.;
          mat[3] = 2./16.;
          mat[4] = 4./16.;
          mat[5] = 2./16.;
          mat[6] = 1./16.;
          mat[7] = 2./16.;
          mat[8] = 1./16.;
      }
      else {
          mat[0] = 0.;
          mat[1] = 0.;
          mat[2] = 0.;
          mat[3] = 0.;
          mat[4] = 1.;
          mat[5] = 0.;
          mat[6] = 0.;
          mat[7] = 0.;
          mat[8] = 0.;
      }
      nThreads = SIZE;
      N = bInfoHeader->imgsize;
      // numero de Blocks en cada dimension 
      uint32_t nBlocksWidth = bInfoHeader->width / nThreads;
      uint32_t nBlocksHeight = bInfoHeader->height / nThreads;

      dim3 dimGrid(nBlocksWidth, nBlocksHeight, 1);
      dim3 dimBlock(nThreads, nThreads, 1);

      hipEventCreate(&E0);
      hipEventCreate(&E1);
      hipEventCreate(&E2);
      hipEventCreate(&E3);

      hipEventRecord(E0, 0);
      hipEventSynchronize(E0);

      hipMalloc((unsigned char**)&imgdata_d, bInfoHeader->imgsize);
      hipMalloc((unsigned char**)&imgdata2_d, bInfoHeader->imgsize);
      hipMalloc((float**)&mat_d, 9 * sizeof(float));
      CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);

      hipMemcpy(imgdata_d, imgdata_h, bInfoHeader->imgsize, hipMemcpyHostToDevice);
      hipMemcpy(imgdata2_d, imgdata_h, bInfoHeader->imgsize, hipMemcpyHostToDevice);
      hipMemcpy(mat_d, mat, 9 * sizeof(float), hipMemcpyHostToDevice);
      CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

      hipEventRecord(E1, 0);
      hipEventSynchronize(E1);
      
      ConvMatKernel<<<dimGrid, dimBlock>>>(imgdata_d, imgdata2_d, bInfoHeader->width, bInfoHeader->height, j, mat_d);
      CheckCudaError((char *) "Invocar Kernel", __LINE__);

      hipEventRecord(E2, 0);
      hipEventSynchronize(E2); 

      hipMemcpy(imgdata_h, imgdata2_d, bInfoHeader->imgsize, hipMemcpyDeviceToHost);
      CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

      hipEventRecord(E3, 0);
      hipEventSynchronize(E3);
      hipEventElapsedTime(&TiempoTotal,  E0, E3);
      hipEventElapsedTime(&TiempoKernel, E1, E2);
      printf("\nKERNEL BlackAndWhiteFilter\n");
      printf("Dimensiones: %d\n",N);
      printf("nThreads: %dx%d (%d)\n", nThreads, nBlocks, nThreads * nBlocks);
      printf("nBlocks: %d\n", nBlocks);
      printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
      printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
      printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoTotal));
      printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));

      hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
      hipFree(imgdata_d);
      hipFree(imgdata2_d);
      hipFree(mat_d);
  }

  /* Cerramos el fichero */
  fclose(f);
  
  printf("el resultado se encuentra en el fichero aux.bmp");

  /* Devolvemos la imagen */
  return imgdata_h;
  
}

void SaveBMP(char *filename, bmpInfoHeader *InfoHeader, unsigned char *imgdata) {
  bmpFileHeader header;
  FILE *f;
  uint16_t type;
  
  f=fopen(filename, "w+");

  header.size = InfoHeader->imgsize + sizeof(bmpFileHeader) + sizeof(bmpInfoHeader) + 2;
  header.resv1 = 0; 
  header.resv2 = 0; 
  /* El offset será el tamaño de las dos cabeceras + 2 (información de fichero)*/
  header.offset=sizeof(bmpFileHeader)+sizeof(bmpInfoHeader)+2;

  /* Escribimos la identificación del archivo */
  type=0x4D42;
  fwrite(&type, sizeof(type),1,f);

  /* Escribimos la cabecera de fichero */
  fwrite(&header, sizeof(bmpFileHeader),1,f);

  /* Escribimos la información básica de la imagen */
  fwrite(InfoHeader, sizeof(bmpInfoHeader),1,f);
  /* Escribimos la imagen */
  printf("%f \n",InfoHeader->imgsize);
  fwrite(imgdata, InfoHeader->imgsize, 1, f);
  fclose(f);
}


//kernel Function, para esta función, necesitamos tanto la fila como la columna en la que actuará nuestro thread en cuestion.
//Ademas tenemos que tener dos copias de imgdata, en uno tendramos los nuevos datos, y la otra la utilizaremos para calcular la primera.


void DisplayInfo(char *FileName, bmpInfoHeader *InfoHeader)
{
  printf("\n");
  printf("Informacion de %s\n", FileName);
  printf("Tamaño de la cabecera: %u bytes\n", InfoHeader->headersize);
  printf("Anchura:               %d pixels\n", InfoHeader->width);
  printf("Altura:                %d pixels\n", InfoHeader->height);
  printf("Planos (1):            %d\n", InfoHeader->planes);
  printf("Bits por pixel:        %d\n", InfoHeader->bpp);
  printf("Compresion:            %d\n", InfoHeader->compress);
  printf("Tamaño de la imagen:   %u bytes\n", InfoHeader->imgsize);
  printf("Resolucion horizontal: %u px/m\n", InfoHeader->bpmx);
  printf("Resolucion vertical:   %u px/m\n", InfoHeader->bpmy);
  if (InfoHeader->bpmx == 0) 
    InfoHeader->bpmx = (unsigned) ((double)24*100/2.54);
  if (InfoHeader->bpmy == 0) 
    InfoHeader->bpmy = (unsigned) ((double)24*100/2.54);

  printf("Colores en paleta:     %d\n", InfoHeader->colors);
  printf("Colores importantes:   %d\n", InfoHeader->imxtcolors);
}

int main(int argc, char** argv) {

  bmpInfoHeader header;

  unsigned char *image;
  
  //int i;
  
  printf("introduce numero del 1 al 4\n1-blur\n2-black and white filter\n3-matriz de convolucion\n4-exit\n");
  
  //scanf("%d", &i);

  image = LoadBMP("./canicas.bmp", &header, 3);
  //DisplayInfo("./canicas.bmp", &header);

  SaveBMP("./auxmat.bmp", &header, image);

}



